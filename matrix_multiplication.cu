#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

// Global Definitions
#define ELEMS 20
#define max 100
#define THREADS_PER_BLOCK 512
#define THREADS_PER_WARP 32

using namespace std;

// Scalar multiplication :: Has many drawbacks
__global__ void scalar_mul(long long *num_row,const long long *ptr,const int *indices, const long long *data , const long long *x, long long *y) {
    int row =  blockDim.x * blockIdx.x + threadIdx.x;
    if(row < *num_row) {
        long long sum_prod = 0.0;
        long long st = ptr[row] ;
        long long end = ptr[row+1];
        for(long long j = st; j < end; j++) {
            sum_prod+= data[j] * x[indices[j]];
        }
        y[row] += sum_prod;
    }
}

// Improved version :: Better use of sequential addressing in warps
__global__ void vector_mul(long long *num_row,const long long *ptr,const int *indices, const long long *data , const long long *x, long long *y) {
    __shared__ long long values[THREADS_PER_BLOCK]; 
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int warp = tid / THREADS_PER_WARP;
    int row = warp;
    int warp_tid = tid % 32;

    if(row < *num_row) {
        long long st = ptr[row] ;
        long long end = ptr[row+1] ;
        values[threadIdx.x] = 0.0;
        
        for(long long j = st + warp_tid; j < end; j += 32)
            values[threadIdx.x] += data[j] * x[indices[j]];
        
        if(warp_tid<16) values[threadIdx.x] += values[threadIdx.x+16]; 
        if(warp_tid<8) values[threadIdx.x] += values[threadIdx.x+8];
        if(warp_tid<4) values[threadIdx.x] += values[threadIdx.x+4];
        if(warp_tid<2) values[threadIdx.x] += values[threadIdx.x+2]; 
        if(warp_tid<1) values[threadIdx.x] += values[threadIdx.x+1];    

        if(warp_tid == 0 ) {    
            y[row] += values[threadIdx.x];
        }
    }

}

int main(int argc, char *argv[]) {
	MPI_Init(&argc,&argv);
    
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    
    int my_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &my_rank);

    long long *ptr;
    vector<int> column;
    vector<long long> data;
    long long *vec;
    long long size;

    // Read input and initialize
    if(my_rank == 0) {
        fstream file;
        file.open(argv[1]);
        char temp[100];
        file >> temp;
        char name[100];
        file >> name;
        file >> temp;
        file >> size;

        ptr = new long long[size+1];
        file >> temp;
        char input[20];

        ptr[0] = 0;
        long long old = 0;
        long long i = 0;
        while(!file.eof()) {
            file >> input;
            string input_str(input);
            if(input_str.compare("B") == 0) 
                break;
            long long row = atoll(input);
            if(row != old) {
            	for(long long j = old+1; j<row; j++)
            		ptr[j] = i;
          		ptr[row] = i;
          		old = row;
            }
            file >> input;
            int col = atoi(input);
            column.push_back(col);
            file >> input;
            long long dat = atoll(input);
            data.push_back(dat);
            i++;	 
        }
        
        ptr[size] = i; 
        long long index;
        vec = new long long[size];
        i = 0;
        while(!file.eof()) {
            file >> input;
            index = atoll(input);
            vec[i]= index;
            i++;
        }
    }    
       
    // Input, Output done
    // Time to distribute the data
    long long new_size,dat_size;
    vector<long long> process_vec;
    vector<int> process_col;
    vector<long long> process_dat;
    long long *process_ptr ;
    long long *row_st = new long long[world_size];
    long long *row_end = new long long[world_size];
    long long *val_start = new long long[world_size];
    long long *val_end = new long long[world_size];
    long long *new_size1 = new long long[world_size];
    long long *dat_size1 = new long long[world_size];
    
    // Distribute Data
    if(my_rank == 0) {         
        long long total = ptr[size];
        long long st1 = 0, end1 = total/world_size;
        int k = 0;
        for(long long i = 0;i<size;i++) {
            if(ptr[i]>=st1) {
                row_st[k] = i;
                val_start[k] = ptr[i];
                while(i<size && ptr[i]<end1) {
                    i++;
                }
                row_end[k] = i;
                val_end[k] = ptr[i];
                new_size1[k] = row_end[k] - row_st[k];
                dat_size1[k] = val_end[k] - val_start[k];
                k++;
                i--;
                st1 = (k*total)/world_size;
                end1 = ((k+1)*total)/world_size;
            }
        } 
        
        new_size = new_size1[0];
        dat_size = dat_size1[0];
        
        process_vec.resize(size);
        memcpy(&process_vec[0],vec,size*sizeof(long long));
        process_col.resize(dat_size);
        process_dat.resize(dat_size);
        memcpy(&process_col[0],&column[0],dat_size*sizeof(int));
        memcpy(&process_dat[0],&data[0],dat_size*sizeof(long long));
        process_ptr = new long long[new_size1[0]+1];
        memcpy(process_ptr,ptr,(new_size1[0]+1)*sizeof(long long));    
        
        for(int j = 1;j<world_size;j++) {
            cout<<"Sending to "<<j<<", "<<row_st[j]<<","<<val_start[j]<<endl;
            MPI_Send(&new_size1[j],1,MPI_LONG_LONG,j,2,MPI_COMM_WORLD);
            MPI_Send(&dat_size1[j],1,MPI_LONG_LONG,j,3,MPI_COMM_WORLD);
            MPI_Send(&ptr[row_st[j]],(new_size1[j]+1),MPI_LONG_LONG,j,4,MPI_COMM_WORLD);
            MPI_Send(&size,1,MPI_LONG_LONG,j,5,MPI_COMM_WORLD);
            MPI_Send(&vec[0],size,MPI_LONG_LONG,j,6,MPI_COMM_WORLD);
            MPI_Send(&column[val_start[j]],dat_size1[j],MPI_INT,j,7,MPI_COMM_WORLD);
            MPI_Send(&data[val_start[j]],dat_size1[j],MPI_LONG_LONG,j,8,MPI_COMM_WORLD);
        }
    }
    // Receive Data
    else {
        cout<<"Rank rec "<<my_rank<<endl;
        MPI_Recv(&new_size,1,MPI_LONG_LONG,0,2,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        MPI_Recv(&dat_size,1,MPI_LONG_LONG,0,3,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        process_ptr = new long long[new_size+1];
        process_col.resize(dat_size);
        process_dat.resize(dat_size);
        MPI_Recv(process_ptr,new_size+1,MPI_LONG_LONG,0,4,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        MPI_Recv(&size,1,MPI_LONG_LONG,0,5,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        process_vec.resize(size);
        MPI_Recv(&process_vec[0],size,MPI_LONG_LONG,0,6,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        MPI_Recv(&process_col[0],dat_size,MPI_INT,0,7,MPI_COMM_WORLD,MPI_STATUS_IGNORE);
        MPI_Recv(&process_dat[0],dat_size,MPI_LONG_LONG,0,8,MPI_COMM_WORLD,MPI_STATUS_IGNORE);     
        cout<<"Rank received "<<my_rank<<endl;        
    }

    // Distribution done
    long long old_ptr = process_ptr[0];
    for(long long j =0;j<new_size+1;j++) {
        process_ptr[j] = process_ptr[j] - old_ptr;
    }    
    cout<<"Rank here is "<<my_rank<<" :: "<<new_size<<","<<dat_size<<","<<endl;     
    long long *num_row;
    long long *host_ans = new long long[new_size];
    long long *zero_ans = new long long[new_size];
    memset(zero_ans,0.0,new_size*sizeof(long long));
    
    int  *gpu_col;
    long long *gpu_ptr;
    long long *gpu_vec,*gpu_dat , *gpu_ans;

    // Allocate CUDA memory
    hipMalloc((void **)&gpu_vec,size*sizeof(long long));
    hipMalloc((void **)&gpu_ans,new_size*sizeof(long long));
    hipMalloc((void **)&gpu_col,dat_size*sizeof(int));
    hipMalloc((void **)&gpu_dat,dat_size*sizeof(long long));
    hipMalloc((void **)&gpu_ptr,(new_size+1)*sizeof(long long));
    hipMalloc((void **)&num_row,sizeof(long long));

    // Initialize CUDA memory
    hipMemcpy(gpu_col,&process_col[0],dat_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dat,&process_dat[0],dat_size*sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(num_row,&new_size,sizeof(long long),hipMemcpyHostToDevice);
    hipMemcpy(gpu_vec,&process_vec[0],size*sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr,&process_ptr[0],(new_size+1)*sizeof(long long), hipMemcpyHostToDevice);
    hipMemcpy(gpu_ans,zero_ans,(new_size)*sizeof(long long), hipMemcpyHostToDevice);
    
    int num_block;

    if(new_size%16==0)
        num_block = new_size/16;
    else
        num_block = new_size/16 + 1;

    // Run CUDA multiplication
    vector_mul<<<num_block,THREADS_PER_BLOCK>>>(num_row,gpu_ptr,gpu_col,gpu_dat,gpu_vec,gpu_ans);
    hipMemcpy(host_ans,gpu_ans,new_size*sizeof(long long), hipMemcpyDeviceToHost);
       
    MPI_Status status;
    vector<long long> tempo;
    long long size_ans;

    // Receive the segmented output and print to a file
    if(my_rank == 0) {
        ofstream myfile;
        myfile.open(argv[2]);
        for(long long i = 0; i < new_size; i++) {
            myfile << host_ans[i] << "\n";
        }
        for(int i = 1; i < world_size; i++) {
            MPI_Recv(&size_ans, 1, MPI_LONG_LONG, i, 1, MPI_COMM_WORLD, &status);
            tempo.resize(size_ans);
            MPI_Recv(&tempo[0], size_ans, MPI_LONG_LONG, i, 1, MPI_COMM_WORLD, &status);
            for(long long j = 0; j < size_ans; j++) {
                myfile << tempo[j] << "\n";
            }
        }
        myfile.close();
    }
    else {
        MPI_Send(&new_size, 1, MPI_LONG_LONG, 0, 1, MPI_COMM_WORLD);
        MPI_Send(&host_ans[0], new_size, MPI_LONG_LONG, 0, 1, MPI_COMM_WORLD);
    }

    // Free CUDA memory
    hipFree(num_row);
    hipFree(gpu_col);
    hipFree(gpu_dat);
    hipFree(gpu_ans);
    hipFree(gpu_ptr);
    hipFree(gpu_vec);
    MPI_Finalize();

}